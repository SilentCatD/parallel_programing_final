#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdint.h>

// !! Not working when there're multiple block doing constructMinCostTable | DEADLOCK

// - Use shared memory and constant memory in 2 somber convo steps
// - Use cuda stream to parallel 2 somber convo step
// - cross-block communication using atomicAdd (not working)

#define FILTER_WIDTH 3
__constant__ int dc_xSombelFilter[FILTER_WIDTH * FILTER_WIDTH];
__constant__ int dc_ySombelFilter[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);                                                                 
		hipEventSynchronize(start);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

__global__ void convoImageKernel(unsigned char* inPixels, int width, int height,
        int filterWidth, int * outPixels, bool isXSombel = true)
{
	extern __shared__ unsigned char s_inPixels[];

	int sharedWidth = blockDim.x + filterWidth - 1;
	int filterRadius = filterWidth / 2;


	// Copy batch 1
	int dest = threadIdx.y * blockDim.x + threadIdx.x;
	int destY = dest / sharedWidth; 
	int destX = dest % sharedWidth;
	int s_inPixelsIdx = destY * sharedWidth + destX;
	int srcY = blockIdx.y * blockDim.y + destY - filterRadius;
	int srcX = blockIdx.x * blockDim.x + destX - filterRadius;

	srcY = min(max(0, srcY), height - 1);
	srcX = min(max(0, srcX), width - 1);
	
	int srcIdx = srcY * width + srcX;
	s_inPixels[s_inPixelsIdx] = inPixels[srcIdx];


	// Copy batch 2
	dest =  threadIdx.y * blockDim.x + threadIdx.x +  blockDim.x * blockDim.y;
	destY = dest / sharedWidth;
	destX = dest % sharedWidth;
	s_inPixelsIdx = destY * sharedWidth + destX;
	srcY = blockIdx.y * blockDim.y + destY - filterRadius;
	srcX = blockIdx.x * blockDim.x + destX - filterRadius;

	srcY = min(max(0, srcY), height - 1);
	srcX = min(max(0, srcX), width - 1);

	srcIdx = srcY * width + srcX; 
	if(destY < sharedWidth){
		s_inPixels[s_inPixelsIdx] = inPixels[srcIdx];
	}
	__syncthreads();
	// convo
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (r < height && c < width){

		int *filter;
		if(isXSombel){
			filter = dc_xSombelFilter;
		}else{
			filter = dc_ySombelFilter;
		}

		int outPixel = 0;
		for (int filterR = 0; filterR < filterWidth; filterR++){
			for(int filterC = 0; filterC < filterWidth; filterC++){
				unsigned char s_inPixel = s_inPixels[(threadIdx.y + filterR) * sharedWidth + (threadIdx.x + filterC)];
				int filterVal =  filter[filterR * filterWidth + filterC];
				outPixel += s_inPixel * filterVal; 
			}
		}
		outPixels[r * width + c] = outPixel;	
	}
}							


__global__ void energyCalcKernel(int* xSombelOut, int* ySombelOut, int width, int height, int *outPixels){
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (c < width && r < height) {
		int i = r * width + c;
		outPixels[i] = abs(xSombelOut[i]) + abs(ySombelOut[i]);
	}

}

__global__ void convertRgb2GrayKernel(unsigned char * inPixels, int width, int height, 
		unsigned char * outPixels)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (c < width && r < height) {
		int i = r * width + c;
		unsigned char red = inPixels[3 * i];
		unsigned char green = inPixels[3 * i + 1];
		unsigned char blue = inPixels[3 * i + 2];
		outPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
	}
}

__device__ int bCount = 0;
__global__ void constructEnergyCostPathTableKernel(int* energy, int width, int height, int* costTable, int*pathTable){
	int cIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if(cIdx< width){
		costTable[cIdx] = energy[cIdx];
		pathTable[cIdx] = 0;
	}
	__syncthreads();

	if(cIdx < width){
		for(int r = 1; r < height; r++){

			int minRIdx = r - 1;

			int leftC = max(0, cIdx - 1);
			int rightC = min(width - 1, cIdx + 1);

			int leftVal = costTable[minRIdx * width + leftC];
			int midVal = costTable[minRIdx * width + cIdx];
			int rightVal = costTable[minRIdx * width + rightC];

			int minVal = min(leftVal, min(midVal, rightVal));

			int idx = r * width + cIdx;
			int costVal = minVal + energy[idx];
			costTable[idx] = costVal;
			if(minVal == leftVal){
				if(leftC == threadIdx.x){
					pathTable[idx] = 0;
				}else{
					pathTable[idx] = -1;
				}
			}else if(minVal == midVal){
				pathTable[idx] = 0;
			}else{
				if(rightC == threadIdx.x){
					pathTable[idx] = 0;
				}else{
					pathTable[idx] = 1;
				}
			}

			if(threadIdx.x == 0){
				atomicAdd(&bCount ,1);
				__threadfence();
				while(bCount < r * gridDim.x){}
			}
			__syncthreads();
		}
	}

}

__global__ void findMinCIdxKernel(int *costTable, int width, int height, int sharedArrMemSize, int* localMinIdx, int* localMin){
	extern __shared__ int s_mem[];
	int* s_costVal = (int*) s_mem;	
	int* s_costIdx = (int *) &(s_mem[sharedArrMemSize]);

	int lastRowIdx = (height - 1) * width;
	
	// Load data to shared mem
	int i1 = 2 * blockDim.x * blockIdx.x + threadIdx.x;
	int i2 = i1 +  blockDim.x;

	if(i1 < width){
		s_costVal[threadIdx.x] = costTable[lastRowIdx + i1];
		s_costIdx[threadIdx.x] = i1;
	}
	if(i2 < width){
		s_costVal[threadIdx.x + blockDim.x] = costTable[lastRowIdx + i2];
		s_costIdx[threadIdx.x + blockDim.x] = i2;
	}
	__syncthreads();

	for(int stride = blockDim.x; stride > 0; stride/=2){
		if(threadIdx.x < stride){
			int value1 = s_costVal[threadIdx.x];
			int value2 = s_costVal[threadIdx.x + stride];
			int index1= s_costIdx[threadIdx.x];
			int index2 = s_costIdx[threadIdx.x + stride];
			if(index1 < width && index2 < width && value2 < value1){
				s_costVal[threadIdx.x] = value2;
				s_costIdx[threadIdx.x] = index2;
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0){
		localMinIdx[blockIdx.x] = s_costIdx[0];
		localMin[blockIdx.x] = s_costVal[0];
	}

}

void findSeam(int minCIdx, int* pathTable, int width, int height, int* seamPos){
	for(int r = height - 1; r >= 0; r--){
		seamPos[r * 2] = r;		
		seamPos[r * 2 + 1] = minCIdx;		

		int nextC = pathTable[r * width + minCIdx];
		if(nextC == -1){
			minCIdx --;

		}else if(nextC == 1){
			minCIdx++;
		}
	}
}

__global__ void addSeamOnDevice(unsigned char* inPixels, unsigned char* inPixels1, int width, int height, int* deviceSeamPos)
{
	// do stuff
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	if (c < width && r < height) {
		int i = r * width + c;
		if (c < deviceSeamPos[2 * r + 1]) {
			inPixels1[3 * i] = inPixels[3 * i];
			inPixels1[3 * i + 1] = inPixels[3 * i + 1];
			inPixels1[3 * i + 2] = inPixels[3 * i + 2];
		}
		else if (c == deviceSeamPos[2 * r + 1]) {		
			int o = r * width + c + 1;
			inPixels1[3 * i] = inPixels[3 * i];
			inPixels1[3 * i + 1] = inPixels[3 * i + 1];
			inPixels1[3 * i + 2] = inPixels[3 * i + 2];
			inPixels1[3 * o] = inPixels[3 * i];
			inPixels1[3 * o + 1] = inPixels[3 * i + 1];
			inPixels1[3 * o + 2] = inPixels[3 * i + 2];
		}
		else {	
			int o = r * width + c + 1;
			inPixels1[3 * o] = inPixels[3 * i];
			inPixels1[3 * o + 1] = inPixels[3 * i + 1];
			inPixels1[3 * o + 2] = inPixels[3 * i + 2];
		}
	}
}

void findSeamOnDeivce(unsigned char* inPixels, int& width, int height, int* deviceSeamPos, int* outCostTable, int* outPathTable, int &outMinColIdx, dim3 convoBlockSize = dim3(1, 1), int costTableBlockSize = 1024, int minColIdxBlockSize = 512){

	// Allocate memory
	GpuTimer timer;
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("GPU name: %s\n", devProp.name);
	printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

	unsigned char* d_inPixels;
	size_t nBytes = width * height * sizeof(unsigned char);
	CHECK(hipMalloc(&d_inPixels, nBytes*3));
	CHECK(hipMemcpy(d_inPixels, inPixels, nBytes * 3, hipMemcpyHostToDevice));
	dim3 gridSize((width - 1) / convoBlockSize.x + 1, (height - 1) / convoBlockSize.y + 1);
	int costTableGridSize = (width - 1) / costTableBlockSize + 1;
	int minColIdxGridSize  = (width - 1)/ (minColIdxBlockSize * 2) + 1; 
	int minColIdxSharedDataSize = minColIdxBlockSize * 4 * sizeof(int);

	// grayScale
	unsigned char *d_outGrayScale;
	CHECK(hipMalloc(&d_outGrayScale, nBytes));
	

	// Sombel
	int s_inPixelsSize = ((convoBlockSize.x + FILTER_WIDTH - 1) * (convoBlockSize.y + FILTER_WIDTH - 1)) * sizeof(unsigned char);

	// xSombel
	int xSombelFilter[] = {1, 0, -1, 2, 0, -2, 1, 0, -1}; 
	int *d_xSombelOut;
    hipStream_t xSombelConvoStream;
    CHECK(hipStreamCreate(&xSombelConvoStream));
	CHECK(hipMalloc(&d_xSombelOut, width * height * sizeof(int)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_xSombelFilter), xSombelFilter, sizeof(xSombelFilter)));

	// ySombel	
	int ySombelFilter[] = {1, 2, 1, 0, 0, 0, -1, -2, -1}; 
	int  * d_ySombelOut;
    hipStream_t ySombelConvoStream;
    CHECK(hipStreamCreate(&ySombelConvoStream));
	CHECK(hipMalloc(&d_ySombelOut, width * height * sizeof(int)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_ySombelFilter), ySombelFilter, sizeof(ySombelFilter)));

	// energy
	int* d_energy;
	CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));

	// construct energy cost table
	int* d_costTable, *d_pathTable;
	CHECK(hipMalloc(&d_costTable, width * height * sizeof(int)));
	CHECK(hipMalloc(&d_pathTable, width * height * sizeof(int)));

	// find min idx
	int* d_localMinIdx,* d_localMin;
	CHECK(hipMalloc(&d_localMinIdx, minColIdxGridSize * sizeof(int)));
	CHECK(hipMalloc(&d_localMin, minColIdxGridSize * sizeof(int)));
	int * localMin = (int*)malloc(minColIdxGridSize * sizeof(int));
	int* localMinIdx = (int*) malloc(minColIdxGridSize * sizeof(int));

	// Execute
	timer.Start();

	// grayScale
	convertRgb2GrayKernel<<<gridSize, convoBlockSize>>>(d_inPixels, width, height, d_outGrayScale);
	
	hipDeviceSynchronize();
	// xSombel
	convoImageKernel<<<gridSize, convoBlockSize, s_inPixelsSize, xSombelConvoStream>>>(d_outGrayScale, width, height, FILTER_WIDTH, d_xSombelOut, true);

	// ySombel	
	convoImageKernel<<<gridSize, convoBlockSize, s_inPixelsSize, ySombelConvoStream>>>(d_outGrayScale, width, height, FILTER_WIDTH, d_ySombelOut, false);
	hipDeviceSynchronize();

	// energy
	energyCalcKernel<<<gridSize, convoBlockSize>>>(d_xSombelOut, d_ySombelOut, width, height, d_energy);

	// construct energy cost table
	constructEnergyCostPathTableKernel<<<costTableGridSize, costTableBlockSize>>>(d_energy, width, height, d_costTable, d_pathTable);

	// find min idx
	findMinCIdxKernel<<<minColIdxGridSize, minColIdxBlockSize, minColIdxSharedDataSize>>>(d_costTable, width, height, minColIdxBlockSize * 2, d_localMinIdx, d_localMin);
	CHECK(hipMemcpy(localMin, d_localMin, minColIdxGridSize * sizeof(int), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(localMinIdx, d_localMinIdx, minColIdxGridSize * sizeof(int), hipMemcpyDeviceToHost));

	int minColumn = localMinIdx[0];
	int minVal = localMin[0];
	for(int i = 0 ; i < minColIdxGridSize; i++){
		if(localMin[i] < minVal){
			minVal = localMin[i];
			minColumn = localMinIdx[i];
		}
	}

	CHECK(hipMemcpy(outPathTable, d_pathTable, width * height * sizeof(int), hipMemcpyDeviceToHost));
	findSeam(minColumn, outPathTable, width, height, deviceSeamPos);
	
	hipDeviceSynchronize();

	hipDeviceSynchronize();
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time of device: %f ms\n\n", time);

	CHECK(hipMemcpy(outCostTable, d_costTable, width * height * sizeof(int), hipMemcpyDeviceToHost));
	outMinColIdx = minColumn;

	// add seam
	int* d_SeamPos;
	CHECK(hipMalloc(&d_SeamPos, height * sizeof(int) * 2));
	CHECK(hipMemcpy(d_SeamPos, deviceSeamPos, height * sizeof(int) * 2, hipMemcpyHostToDevice));
	unsigned char* d_inPixels1;
	for (int num = 0; num < 50; num++) {
		CHECK(hipMalloc(&d_inPixels1, (width + 1) * height * sizeof(unsigned char) * 3))
		addSeamOnDevice<<<gridSize, convoBlockSize>>>(d_inPixels, d_inPixels1, width, height, d_SeamPos);
		CHECK(hipFree(d_inPixels));
		d_inPixels = d_inPixels1;
		width++;
	}

	CHECK(hipFree(d_inPixels));
	CHECK(hipFree(d_outGrayScale));
	CHECK(hipFree(d_xSombelOut));
	CHECK(hipFree(d_ySombelOut));
	CHECK(hipFree(d_energy));
	CHECK(hipFree(d_costTable));
	CHECK(hipFree(d_pathTable));
    CHECK(hipStreamDestroy(xSombelConvoStream));
    CHECK(hipStreamDestroy(ySombelConvoStream));
}

void convertRgb2Gray(unsigned char * inPixels, int width, int height,
		unsigned char * outPixels)
{
	for (int r = 0; r < height; r++) {
		for (int c = 0; c < width; c++)
		{
			int i = r * width + c;
			unsigned char red = inPixels[3 * i];
			unsigned char green = inPixels[3 * i + 1];
			unsigned char blue = inPixels[3 * i + 2];
			outPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
		}
	}
}


void convoImage(unsigned char * inPixels, int width, int height, int * filter, int filterWidth, 
        int* outPixels)
{
	for (int outPixelsR = 0; outPixelsR < height; outPixelsR++)
	{
		for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
		{
			int outPixel = 0;
			for (int filterR = 0; filterR < filterWidth; filterR++)
			{
				for (int filterC = 0; filterC < filterWidth; filterC++)
				{
					int filterVal = filter[filterR*filterWidth + filterC];
					int inPixelsR = outPixelsR - filterWidth/2 + filterR;
					int inPixelsC = outPixelsC - filterWidth/2 + filterC;
					inPixelsR = min(max(0, inPixelsR), height - 1);
					inPixelsC = min(max(0, inPixelsC), width - 1);
					unsigned char inPixel = inPixels[inPixelsR*width + inPixelsC];
					outPixel += filterVal * inPixel;
				}
			}
			outPixels[outPixelsR*width + outPixelsC] = outPixel; 
		}
	}
}

void energyCalc(int* xSombelOut, int* ySomberOut, int width, int height, int* outPixels){
	for(int i = 0; i < width * height; i++){
		outPixels[i] = abs(xSombelOut[i]) + abs(ySomberOut[i]);
	}
}

void constructEnergyCostPathTable(int *energy, int width, int height, int* costTable,int* pathTable){
	for(int r = 0; r < height; r++){
		for(int c = 0; c < width; c++){
			if(r == 0){
				costTable[c] = energy[c];
				pathTable[c] = 0;
			}else{
				int idx = r * width + c;

				int minRIdx = r - 1;
				
				int leftC = max(0, c - 1);

				int rightC = min(width - 1, c + 1);

				int leftVal = costTable[minRIdx * width + leftC];
				int midVal = costTable[minRIdx * width + c];
				int rightVal = costTable[minRIdx * width + rightC];

				int minVal = min(leftVal, min(midVal, rightVal));

				costTable[idx] = minVal + energy[idx];
				if(minVal == leftVal){
					if(leftC == c){
						pathTable[idx] = 0;
					}else{
						pathTable[idx] = -1;
					}
				}else if(minVal == midVal){
					pathTable[idx] = 0;
				}else{
					if(rightC == c){
						pathTable[idx] = 0;
					}else{
						pathTable[idx] = 1;
					}
				}

			}
		}
	}
}

int findMinCIdx(int* costTable, int width, int height){
	int idx = (height -1) * width;
	int result = 0;
	int currentMin = costTable[idx];
	for(int i = 0; i < width; i++){
		if(costTable[idx + i] <  currentMin){
			currentMin = costTable[idx + i];
			result = i;
		}
	}
	return result;
}


void findSeamOnHost(unsigned char* inPixels, int width, int height, int* seamPos, int * outCostTable, int* outPathTable, int &outMinColIdx){

	// Allocate memory
	GpuTimer timer;
	// grayScale
	unsigned char *grayScale = (unsigned char*) malloc(width * height * sizeof(unsigned char));
	// xSomber
	int xSombelFilter[] = {1, 0, -1, 2, 0, -2, 1, 0, -1}; 
	int* xSombelOut = (int*) malloc(width * height * sizeof(int));
	// ySomber	
	int ySombelFilter[] = {1, 2, 1, 0, 0, 0, -1, -2, -1}; 
	int* ySombelOut = (int*) malloc(width * height *  sizeof(int));
	// energy
	int *energy = (int*) malloc(width * height * sizeof(int));

	// construct energy cost table
	int *costTable = (int*) malloc(width * height * sizeof(int));
	int *pathTable = (int*) malloc(width * height * sizeof(int));

	// Execute
	timer.Start();
	// grayScale
	convertRgb2Gray(inPixels,width, height, grayScale);

	// xSomber
	convoImage(grayScale, width, height, xSombelFilter, 3, xSombelOut);

	// ySomber	
	convoImage(grayScale, width, height, ySombelFilter, 3, ySombelOut);

	// energy
	energyCalc(xSombelOut, ySombelOut, width, height, energy);

	// construct energy cost table
	constructEnergyCostPathTable(energy, width, height, costTable, pathTable);

	// find min column
	int minColumn = findMinCIdx(costTable, width, height);

	// find seam
	findSeam(minColumn, pathTable, width, height, seamPos);

	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time of host: %f ms\n\n", time);

	for(int i = 0; i < width * height; i++){
		outCostTable[i] = costTable[i];
		outPathTable[i] = pathTable[i];
	}
	outMinColIdx = minColumn;

	free(grayScale);
	free(costTable); 
	free(pathTable);
	free(xSombelOut);
	free(ySombelOut);
	free(energy);
}


void readPnm(char * fileName, int &width, int &height, unsigned char * &pixels)
{
	FILE * f = fopen(fileName, "rb");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P6") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	int c = getc(f);
    while (c == '#') {
    while (getc(f) != '\n') ;
         c = getc(f);
    }
    ungetc(c, f);

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}
	size_t nBytes = width * height * 3 * sizeof(unsigned char);
	pixels = (unsigned char*)malloc(nBytes);
	fread(pixels, nBytes, 1, f);
	fclose(f);
}

void writePnm(unsigned char* pixels, int numChannels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "wb");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	
	if (numChannels == 1)
		fprintf(f, "P5\n");
	else if (numChannels == 3)
		fprintf(f, "P6\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i %i\n255", width, height); 
	size_t nBytes = width * height * numChannels * sizeof(unsigned char);

	fwrite(pixels, nBytes, 1, f);
	
	fclose(f);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

double checkCorrect(int* out, int* out2, int width, int height){
	float err = 0;
	int n =  width * height;
	for (int i = 0; i < n; i++)
		err += abs(out[i] - out2[i]);
	err /= n;
	return err;
}

double checkCorrectPos(int* out, int *out2, int height){
	float err = 0;
	for(int i = 0; i < height; i++){
		err += abs(out[i*2] - out2[i*2]) + abs(out[i * 2 + 1] - out2[i * 2 + 1]);
	}
	return err / (height);
}

int main(int argc, char ** argv)
{

	// Read input image file
	int width, height;
	unsigned char * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("\nImage size (width x height): %i x %i\n", width, height);

	int *hostSeamPos = (int*) malloc(height * 2 * sizeof(int));
	int *outCostTableHost = (int*) malloc(width * height * sizeof(int));
	int *outPathTableHost = (int*) malloc(width * height * sizeof(int));
	int outMinColIdxHost;
	findSeamOnHost(inPixels, width, height, hostSeamPos, outCostTableHost, outPathTableHost, outMinColIdxHost);

	int *deviceSeamPos = (int*) malloc(height * 2 * sizeof(int));
	int *outCostTableDevice = (int*) malloc(width * height * sizeof(int));
	int *outPathTableDevice = (int*) malloc(width * height * sizeof(int));
	int outMinColIdxDevice;
	findSeamOnDeivce(inPixels, width, height,deviceSeamPos ,outCostTableDevice, outPathTableDevice, outMinColIdxDevice,dim3(32, 32), 1024, 256);

	double errCostTable = checkCorrect(outCostTableDevice, outCostTableHost, width, height);
	double errPathTable = checkCorrect(outPathTableDevice, outPathTableHost, width, height);
	double errPos =  checkCorrectPos(hostSeamPos, deviceSeamPos, height);
	printf("Error cost table: %f\n", errCostTable);
	printf("Error path table: %f\n", errPathTable);
	printf("min col host: %d | min col device: %d\n", outMinColIdxHost, outMinColIdxDevice);
	printf("Error seam pos: %f\n", errPos);

	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(outPixels, 1, width, height, concatStr(outFileNameBase, "_device.pnm"));
	// for(int i = 0; i < height; i++){
	// 	printf("host pos: %d, %d | device pos: %d %d\n", hostSeamPos[i*2], hostSeamPos[i*2+1], deviceSeamPos[i*2], deviceSeamPos[i*2 +  1]);
	// }


	free(inPixels);
	free(hostSeamPos);
	free(deviceSeamPos);
	free(outCostTableHost);
	free(outPathTableHost);
	free(outCostTableDevice);
	free(outPathTableDevice);
}
